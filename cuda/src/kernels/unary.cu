
#include <hip/hip_runtime.h>
static __device__ __forceinline__ float op_abs(float x) {
    return fabsf(x);
}

static __device__ __forceinline__ float op_sgn(float x) {
    return (x > 0.f ? 1.f : ((x < 0.f ? -1.f : 0.f)));
}

static __device__ __forceinline__ float op_neg(float x) {
    return -x;
}

static __device__ __forceinline__ float op_step(float x) {
    return x > 0.0f;
}

static __device__ __forceinline__ float op_gelu(float x) {
    const float GELU_COEF_A    = 0.044715f;
    const float SQRT_2_OVER_PI = 0.79788456080286535587989211986876f;

    return 0.5f*x*(1.0f + tanhf(SQRT_2_OVER_PI*x*(1.0f + GELU_COEF_A*x*x)));
}

static __device__ __forceinline__ float op_gelu_quick(float x) {
    const float GELU_QUICK_COEF = -1.702f;

    return x * (1.0f / (1.0f + expf(GELU_QUICK_COEF * x)));
}

static __device__ __forceinline__ float op_silu(float x) {
    return x / (1.0f + expf(-x));
}

static __device__ __forceinline__ float op_tanh(float x) {
    return tanhf(x);
}

static __device__ __forceinline__ float op_relu(float x) {
    return fmaxf(x, 0);
}

static __device__ __forceinline__ float op_sigmoid(float x) {
    return 1.0f / (1.0f + expf(-x));
}

static __device__ __forceinline__ float op_hardsigmoid(float x) {
    return fminf(1.0f, fmaxf(0.0f, (x + 3.0f) / 6.0f));
}

static __device__ __forceinline__ float op_hardswish(float x) {
    return x * fminf(1.0f, fmaxf(0.0f, (x + 3.0f) / 6.0f));
}

static __device__ __forceinline__ float op_exp(float x) {
    return expf(x);
}

static __device__ __forceinline__ float op_sqr(float x) {
    return x * x;
}

static __device__ __forceinline__ float op_sqrt(float x) {
    return sqrtf(x);
}

static __device__ __forceinline__ float op_sin(float x) {
    return sinf(x);
}

static __device__ __forceinline__ float op_cos(float x) {
    return cosf(x);
}

static __device__ __forceinline__ float op_log(float x) {
    return logf(x);
}

template <float (*op)(float), typename T>
static __global__ void unary_op_kernel(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = (T)op((float)x[i]);
}

template <typename T>
static __global__ void silu(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    float el = (float)x[i];
    dst[i] = (T)(el / (1.0f + expf(-el)));
}

extern "C" __global__ void silu(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    float el = (float)x[i];
    dst[i] = (float)(el / (1.0f + expf(-el)));
}