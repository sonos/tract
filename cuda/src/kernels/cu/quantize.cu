#include "hip/hip_runtime.h"
#include "utils.cuh"
#include <cstdint>
#include <hip/hip_fp16.h>

#define QK8_1 32
#define QI8_1 (QK8_1 / (4 * QR8_1))
#define QR8_1 1

#define QK4_0 32
#define QI4_0 (QK4_0 / (4 * QR4_0))
#define QR4_0 2

#define QK8_0 32
#define QI8_0 (QK8_0 / (4 * QR8_0))
#define QR8_0 1

typedef struct {
  half d;                // delta
  uint8_t qs[QK4_0 / 2]; // nibbles / quants
} block_q4_0;

typedef struct {
  half2 ds;
  int8_t qs[QK8_1]; // quants
} block_q8_1;
static_assert(sizeof(block_q8_1) == 2 * sizeof(half) + QK8_1,
              "wrong q8_1 block size/padding");

struct block_q8_1_mmq {
  // The y float data is converted to a data layout that can simply be copied to
  // shared memory as a contiguous block. The y float data is first grouped as
  // blocks of 128 values. These blocks are then treated as individual data
  // values and transposed.
  //
  // To avoid shared memory bank conflicts each block is padded with 16 bytes.
  // This padding is also used to store block scales/partial sums.
  // The scales multiplied with the quantized data are equal to the unquantized
  // values. The partial sums are obtained by summing up a subgroup of the
  // contained values (prior to quantization)
  //     and are only needed for performance reasons.
  half2 ds4[4]; // 1 16 bit scale + 1 16 bit partial sum per 32 values, stored
                // as d0,s0,d1,s1,d2,s2,d3,s3
  int8_t qs[4 * QK8_1]; // 128 values quantized to 8 bit each
};
static_assert(sizeof(block_q8_1_mmq) == 4 * QK8_1 + 4 * sizeof(half2),
              "Unexpected block_q8_1_mmq size");

static __device__ __forceinline__ void
compute_mmq_q81_block(float4 xi, int64_t ib, int64_t iqs, block_q8_1_mmq *y) {
  constexpr int vals_per_scale = 32;
  constexpr int vals_per_sum = 32;

  float amax = fabsf(xi.x);
  amax = fmaxf(amax, fabsf(xi.y));
  amax = fmaxf(amax, fabsf(xi.z));
  amax = fmaxf(amax, fabsf(xi.w));

// Exchange max. abs. value between vals_per_scale/4 threads.
#pragma unroll
  for (int offset = vals_per_scale / 8; offset > 0; offset >>= 1) {
    amax = fmaxf(amax, __shfl_xor_sync(0xFFFFFFFF, amax, offset, WARP_SIZE));
  }

  float sum;
  sum = xi.x + xi.y + xi.z + xi.w;

// Calculate sums across vals_per_sum/4 threads.
#pragma unroll
  for (int offset = vals_per_sum / 8; offset > 0; offset >>= 1) {
    sum += __shfl_xor_sync(0xFFFFFFFF, sum, offset, WARP_SIZE);
  }

  float d_inv = (amax > 0.f) ? 127.f / amax : 0.f;
  char4 q;
  q.x = roundf(xi.x * d_inv);
  q.y = roundf(xi.y * d_inv);
  q.z = roundf(xi.z * d_inv);
  q.w = roundf(xi.w * d_inv);

  // Write back 4 int8 values as a single 32 bit value for better memroy
  // bandwidth:
  char4 *yqs4 = (char4 *)y[ib].qs;
  yqs4[iqs / 4] = q;

  if (iqs % 32 != 0) {
    return;
  }

  const float d = (d_inv > 0.0f) ? (1.0f / d_inv) : 0.0f;
  y[ib].ds4[iqs / 32] = make_half2(d, sum);
}

extern "C" __global__ void quantize_mmq_q8_1_fast_nd2(
    const float *__restrict__ x, void *__restrict__ vy, const int64_t k,
    const int64_t in_strides_0, const int64_t in_strides_1,
    const int64_t padded_k) {

  const int64_t i0 = ((int64_t)blockDim.x * blockIdx.y + threadIdx.x) * 4;

  if (i0 >= padded_k) {
    return;
  }

  const int64_t i1 = blockIdx.x;

  const int64_t i00 = i0;
  const int64_t i01 = i1;

  const float4 *x4 = (const float4 *)x;

  const int64_t ib0 =
      blockIdx.z * ((int64_t)gridDim.x * gridDim.y * blockDim.x /
                    QK8_1); // first block of channel
  const int64_t ib = ib0 + (i0 / (4 * QK8_1)) * gridDim.x +
                     blockIdx.x;        // block index in channel
  const int64_t iqs = i0 % (4 * QK8_1); // quant index in block

  // Load 4 floats per thread and calculate max. abs. value between them:
  const float4 xi = i0 < k ? x4[(i01 * in_strides_0 + i00) / 4]
                           : make_float4(0.0f, 0.0f, 0.0f, 0.0f);

  compute_mmq_q81_block(xi, ib, iqs, (block_q8_1_mmq *)vy);
}

extern "C" __global__ void quantize_mmq_q8_1_fast_nd3(
    const float *__restrict__ x, void *__restrict__ vy, const int64_t k,
    const int64_t in_strides_0, const int64_t in_strides_1,
    const int64_t in_strides_2,
    const int out_shape_1, const int64_t padded_k) {

  const int64_t i0 = ((int64_t)blockDim.x * blockIdx.y + threadIdx.x) * 4;

  if (i0 >= padded_k) {
    return;
  }

  const int64_t i1 = blockIdx.x;
  const int64_t i2 = blockIdx.z;

  const int64_t i00 = i0;
  const int64_t i01 = i1;
  const int64_t i02 = i2;

  const float4 *x4 = (const float4 *)x;

  const int64_t ib0 =
      blockIdx.z * ((int64_t)gridDim.x * gridDim.y * blockDim.x /
                    QK8_1); // first block of channel
  const int64_t ib = ib0 + (i0 / (4 * QK8_1)) * out_shape_1 +
                     blockIdx.x;        // block index in channel
  const int64_t iqs = i0 % (4 * QK8_1); // quant index in block

  // Load 4 floats per thread and calculate max. abs. value between them:
  const float4 xi = i0 < k ? x4[(i02 * in_strides_0 +
                                 i01 * in_strides_1 + i00) /
                                4]
                           : make_float4(0.0f, 0.0f, 0.0f, 0.0f);

  compute_mmq_q81_block(xi, ib, iqs, (block_q8_1_mmq *)vy);
}

extern "C" __global__ void quantize_mmq_q8_1_fast_nd4(
    const float *__restrict__ x, void *__restrict__ vy, const int64_t k,
    const int64_t in_strides_0, const int64_t in_strides_1,
    const int64_t in_strides_2, const int64_t in_strides_3,
    const int out_shape_1, const int out_shape_2, const int64_t padded_k) {

  const int64_t i0 = ((int64_t)blockDim.x * blockIdx.y + threadIdx.x) * 4;

  if (i0 >= padded_k) {
    return;
  }

  const int64_t i1 = blockIdx.x;
  const int64_t i2 = blockIdx.z % out_shape_1;
  const int64_t i3 = blockIdx.z / out_shape_1;

  const int64_t i00 = i0;
  const int64_t i01 = i1;
  const int64_t i02 = i2;
  const int64_t i03 = i3;

  const float4 *x4 = (const float4 *)x;

  const int64_t ib0 =
      blockIdx.z * ((int64_t)gridDim.x * gridDim.y * blockDim.x /
                    QK8_1); // first block of channel
  const int64_t ib = ib0 + (i0 / (4 * QK8_1)) * out_shape_2 +
                     blockIdx.x;        // block index in channel
  const int64_t iqs = i0 % (4 * QK8_1); // quant index in block

  // Load 4 floats per thread and calculate max. abs. value between them:
  const float4 xi = i0 < k ? x4[(i03 * in_strides_0 + i02 * in_strides_1 +
                                 i01 * in_strides_2 + i00) /
                                4]
                           : make_float4(0.0f, 0.0f, 0.0f, 0.0f);

  compute_mmq_q81_block(xi, ib, iqs, (block_q8_1_mmq *)vy);
}

extern "C" __global__ void quantize_mmq_q8_1_fast_nd5(
    const float *__restrict__ x, void *__restrict__ vy, const int64_t k,
    const int64_t in_strides_0, const int64_t in_strides_1,
    const int64_t in_strides_2, const int64_t in_strides_3, const int64_t in_strides_4,
    const int out_shape_1, const int out_shape_2, const int out_shape_3, const int64_t padded_k) {

  const int64_t i0 = ((int64_t)blockDim.x * blockIdx.y + threadIdx.x) * 4;

  if (i0 >= padded_k) {
    return;
  }

  const int64_t i1 = blockIdx.x;
  const int64_t i2 = blockIdx.z % out_shape_2;
  const int64_t i3 = (blockIdx.z / out_shape_2) % out_shape_1;
  const int64_t i4 = blockIdx.z / (out_shape_2 * out_shape_1);

  const int64_t i00 = i0;
  const int64_t i01 = i1;
  const int64_t i02 = i2;
  const int64_t i03 = i3;
  const int64_t i04 = i4;

  const float4 *x4 = (const float4 *)x;

  const int64_t ib0 =
      blockIdx.z * ((int64_t)gridDim.x * gridDim.y * blockDim.x /
                    QK8_1); // first block of channel
  const int64_t ib = ib0 + (i0 / (4 * QK8_1)) * out_shape_3 +
                     blockIdx.x;        // block index in channel
  const int64_t iqs = i0 % (4 * QK8_1); // quant index in block

  // Load 4 floats per thread and calculate max. abs. value between them:
  const float4 xi = i0 < k ? x4[(i04 * in_strides_4 + i03 * in_strides_1 +
                                 i02 * in_strides_2 + i01 * in_strides_3 + i00) /
                                4]
                           : make_float4(0.0f, 0.0f, 0.0f, 0.0f);

  compute_mmq_q81_block(xi, ib, iqs, (block_q8_1_mmq *)vy);
}

extern "C" __global__ void
quantize_mmq_q8_1_nd2(const float *__restrict__ x, void *__restrict__ vy,
                      const int64_t k, const int64_t in_strides_0,
                      const int64_t in_strides_1, const int64_t padded_k) {

  const int64_t i0 = ((int64_t)blockDim.x * blockIdx.y + threadIdx.x) * 4;

  if (i0 >= padded_k) {
    return;
  }

  const int64_t i1 = blockIdx.x;

  float4 xi = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
  const int64_t base = i1 * in_strides_0 + i0 * in_strides_1;

  xi.x = (i0 + 0 < k) ? x[base + 0 * in_strides_1] : 0.0f;
  xi.y = (i0 + 1 < k) ? x[base + 1 * in_strides_1] : 0.0f;
  xi.z = (i0 + 2 < k) ? x[base + 2 * in_strides_1] : 0.0f;
  xi.w = (i0 + 3 < k) ? x[base + 3 * in_strides_1] : 0.0f;

  const int64_t ib0 =
      blockIdx.z * ((int64_t)gridDim.x * gridDim.y * blockDim.x /
                    QK8_1); // first block of channel
  const int64_t ib = ib0 + (i0 / (4 * QK8_1)) * gridDim.x +
                     blockIdx.x;        // block index in channel
  const int64_t iqs = i0 % (4 * QK8_1); // quant index in block

  compute_mmq_q81_block(xi, ib, iqs, (block_q8_1_mmq *)vy);
}

extern "C" __global__ void
quantize_mmq_q8_1_nd3(const float *__restrict__ x, void *__restrict__ vy,
                      const int64_t k, const int64_t in_strides_0,
                      const int64_t in_strides_1, const int64_t in_strides_2,
                      const int out_shape_1, const int64_t padded_k) {

  const int64_t i0 = ((int64_t)blockDim.x * blockIdx.y + threadIdx.x) * 4;

  if (i0 >= padded_k) {
    return;
  }

  const int64_t i1 = blockIdx.x;
  const int64_t i2 = blockIdx.z;

  float4 xi = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
  const int64_t base =
      i2 * in_strides_0 + i1 * in_strides_1 + i0 * in_strides_2;

  xi.x = (i0 + 0 < k) ? x[base + 0 * in_strides_2] : 0.0f;
  xi.y = (i0 + 1 < k) ? x[base + 1 * in_strides_2] : 0.0f;
  xi.z = (i0 + 2 < k) ? x[base + 2 * in_strides_2] : 0.0f;
  xi.w = (i0 + 3 < k) ? x[base + 3 * in_strides_2] : 0.0f;

  const int64_t ib0 =
      blockIdx.z * ((int64_t)gridDim.x * gridDim.y * blockDim.x /
                    QK8_1); // first block of channel
  const int64_t ib = ib0 + (i0 / (4 * QK8_1)) * out_shape_1 +
                     blockIdx.x;        // block index in channel
  const int64_t iqs = i0 % (4 * QK8_1); // quant index in block

  compute_mmq_q81_block(xi, ib, iqs, (block_q8_1_mmq *)vy);
}

extern "C" __global__ void
quantize_mmq_q8_1_nd4(const float *__restrict__ x, void *__restrict__ vy,
                      const int64_t k, const int64_t in_strides_0,
                      const int64_t in_strides_1, const int64_t in_strides_2,
                      const int64_t in_strides_3, const int out_shape_1,
                      const int out_shape_2, const int64_t padded_k) {

  const int64_t i0 = ((int64_t)blockDim.x * blockIdx.y + threadIdx.x) * 4;

  if (i0 >= padded_k) {
    return;
  }

  const int64_t i1 = blockIdx.x;
  const int64_t i2 = blockIdx.z % out_shape_1;
  const int64_t i3 = blockIdx.z / out_shape_1;

  float4 xi = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
  const int64_t base = i3 * in_strides_0 + i2 * in_strides_1 +
                       i1 * in_strides_2 + i0 * in_strides_3;

  xi.x = (i0 + 0 < k) ? x[base + 0 * in_strides_3] : 0.0f;
  xi.y = (i0 + 1 < k) ? x[base + 1 * in_strides_3] : 0.0f;
  xi.z = (i0 + 2 < k) ? x[base + 2 * in_strides_3] : 0.0f;
  xi.w = (i0 + 3 < k) ? x[base + 3 * in_strides_3] : 0.0f;

  const int64_t ib0 =
      blockIdx.z * ((int64_t)gridDim.x * gridDim.y * blockDim.x /
                    QK8_1); // first block of channel
  const int64_t ib = ib0 + (i0 / (4 * QK8_1)) * out_shape_2 +
                     blockIdx.x;        // block index in channel
  const int64_t iqs = i0 % (4 * QK8_1); // quant index in block

  compute_mmq_q81_block(xi, ib, iqs, (block_q8_1_mmq *)vy);
}

extern "C" __global__ void
quantize_mmq_q8_1_nd5(const float *__restrict__ x, void *__restrict__ vy,
                      const int64_t k, const int64_t in_strides_0,
                      const int64_t in_strides_1, const int64_t in_strides_2,
                      const int64_t in_strides_3, const int64_t in_strides_4,
                      const int out_shape_1, const int out_shape_2,
                      const int out_shape_3, const int64_t padded_k) {

  const int64_t i0 = ((int64_t)blockDim.x * blockIdx.y + threadIdx.x) * 4;

  if (i0 >= padded_k) {
    return;
  }

  const int64_t i1 = blockIdx.x;
  const int64_t i2 = blockIdx.z % out_shape_2;
  const int64_t i3 = (blockIdx.z / out_shape_2) % out_shape_1;
  const int64_t i4 = blockIdx.z / (out_shape_2 * out_shape_1);

  float4 xi = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
  const int64_t base = i4 * in_strides_0 + i3 * in_strides_1 +
                       i2 * in_strides_2 + i1 * in_strides_3 +
                       i0 * in_strides_4;

  xi.x = (i0 + 0 < k) ? x[base + 0 * in_strides_4] : 0.0f;
  xi.y = (i0 + 1 < k) ? x[base + 1 * in_strides_4] : 0.0f;
  xi.z = (i0 + 2 < k) ? x[base + 2 * in_strides_4] : 0.0f;
  xi.w = (i0 + 3 < k) ? x[base + 3 * in_strides_4] : 0.0f;

  const int64_t ib0 =
      blockIdx.z * ((int64_t)gridDim.x * gridDim.y * blockDim.x /
                    QK8_1); // first block of channel
  const int64_t ib = ib0 + (i0 / (4 * QK8_1)) * out_shape_3 +
                     blockIdx.x;        // block index in channel
  const int64_t iqs = i0 % (4 * QK8_1); // quant index in block

  compute_mmq_q81_block(xi, ib, iqs, (block_q8_1_mmq *)vy);
}

static __device__ __forceinline__ void
compute_q81_block(float xi, int64_t i_cont, block_q8_1 **y_ptr) {
  block_q8_1 *y = *y_ptr;
  const int64_t ib = i_cont / QK8_1;  // block index
  const int64_t iqs = i_cont % QK8_1; // quant index
  float amax = fabsf(xi);
  float sum = xi;

  amax = warp_reduce_max(amax);
  sum = warp_reduce_sum(sum);

  const float d = amax / 127;
  const int8_t q = amax == 0.0f ? 0 : roundf(xi / d);

  y[ib].qs[iqs] = q;

  if (iqs > 0) {
    return;
  }

  reinterpret_cast<half &>(y[ib].ds.x) = d;
  reinterpret_cast<half &>(y[ib].ds.y) = sum;
}

extern "C" __global__ void
quantize_q8_1_nd2(const float *__restrict__ x, void *__restrict__ vy,
                  const int64_t k, const int64_t in_strides_0,
                  const int64_t in_strides_1, const int64_t padded_k) {
  const int64_t i0 = (int64_t)blockDim.x * blockIdx.x + threadIdx.x;

  if (i0 >= padded_k) {
    return;
  }

  const int64_t i1 = blockIdx.y;

  const int64_t &i00 = i0;
  const int64_t &i01 = i1;

  const int64_t i_cont = i1 * padded_k + i0;

  const float xi = i0 < k ? x[i01 * in_strides_0 + i00 * in_strides_1] : 0.0f;

  compute_q81_block(xi, i_cont, (block_q8_1 **)&vy);
}

extern "C" __global__ void
quantize_q8_1_nd3(const float *__restrict__ x, void *__restrict__ vy,
                  const int64_t k, const int64_t in_strides_0,
                  const int64_t in_strides_1, const int64_t in_strides_2,
                  const int64_t out_shape_1, const int64_t padded_k) {
  const int64_t i0 = (int64_t)blockDim.x * blockIdx.x + threadIdx.x;

  if (i0 >= padded_k) {
    return;
  }

  const int64_t i1 = blockIdx.y;
  const int64_t i2 = blockIdx.z;

  const int64_t &i00 = i0;
  const int64_t &i01 = i1;
  const int64_t &i02 = i2;

  const int64_t i_cont = (i2 * out_shape_1 + i1) * padded_k + i0;

  const float xi =
      i0 < k ? x[i02 * in_strides_0 + i01 * in_strides_1 + i00 * in_strides_2]
             : 0.0f;

  compute_q81_block(xi, i_cont, (block_q8_1 **)&vy);
}

extern "C" __global__ void
quantize_q8_1_nd4(const float *__restrict__ x, void *__restrict__ vy,
                  const int64_t k, const int64_t in_strides_0,
                  const int64_t in_strides_1, const int64_t in_strides_2,
                  const int64_t in_strides_3, const int64_t out_shape_1,
                  const int64_t out_shape_2, const int64_t padded_k) {
  const int64_t i0 = (int64_t)blockDim.x * blockIdx.x + threadIdx.x;

  if (i0 >= padded_k) {
    return;
  }

  const int64_t i1 = blockIdx.y;
  const int64_t i2 = blockIdx.z % out_shape_1;
  const int64_t i3 = blockIdx.z / out_shape_1;

  const int64_t &i00 = i0;
  const int64_t &i01 = i1;
  const int64_t &i02 = i2;
  const int64_t &i03 = i3;

  const int64_t i_cont =
      ((i3 * out_shape_1 + i2) * out_shape_2 + i1) * padded_k + i0;

  const float xi = i0 < k ? x[i03 * in_strides_0 + i02 * in_strides_1 +
                              i01 * in_strides_2 + i00 * in_strides_3]
                          : 0.0f;

  compute_q81_block(xi, i_cont, (block_q8_1 **)&vy);
}

extern "C" __global__ void
quantize_q8_1_nd5(const float *__restrict__ x, void *__restrict__ vy,
                  const int64_t k, const int64_t in_strides_4,
                  const int64_t in_strides_3, const int64_t in_strides_2,
                  const int64_t in_strides_1, const int64_t in_strides_0,
                  const int64_t out_shape_1, const int64_t out_shape_2,
                  const int64_t out_shape_3, const int64_t padded_k) {
  const int64_t i0 = (int64_t)blockDim.x * blockIdx.x + threadIdx.x;

  if (i0 >= padded_k) {
    return;
  }

  const int64_t i1 = blockIdx.y;
  const int64_t i2 = blockIdx.z % out_shape_2;
  const int64_t i3 = (blockIdx.z / out_shape_2) % out_shape_1;
  const int64_t i4 = blockIdx.z / (out_shape_2 * out_shape_1);

  const int64_t &i00 = i0;
  const int64_t &i01 = i1;
  const int64_t &i02 = i2;
  const int64_t &i03 = i3;
  const int64_t &i04 = i4;

  const int64_t i_cont =
      (((i4 * out_shape_1 + i3) * out_shape_2 + i2) * out_shape_3 + i1) *
          padded_k +
      i0;

  const float xi =
      i0 < k ? x[i04 * in_strides_0 + i03 * in_strides_1 + i02 * in_strides_2 +
                 i01 * in_strides_3 + i00 * in_strides_4]
             : 0.0f;

  compute_q81_block(xi, i_cont, (block_q8_1 **)&vy);
}